#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <mma.h>

using namespace nvcuda;


__global__ void warpGemm(half *A, half *B, float *C) {
  wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
  wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
  wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;

  wmma::load_matrix_sync(a_frag, A, 16);
  wmma::load_matrix_sync(b_frag, B, 16);
  wmma::fill_fragment(c_frag, 0.0f);

  wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

  wmma::store_matrix_sync(C, c_frag, 16, wmma::mem_row_major);
}

__global__ void float2half_mat(float *a, float *b, int n, half *ha, half *hb) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i < n) {
    ha[i] = __float2half(a[i]);
    hb[i] = __float2half(b[i]);
  }
}

int main() {
    const size_t N2 = 16*16;
    const size_t matSize = sizeof(float) * N2;

    // Generate inputs
    float input_A = [
            7.0,7.0,6.0,1.0,8.0,6.0,6.0,9.0,9.0,6.0,4.0,5.0,5.0,0.0,2.0,6.0
            5.0,6.0,7.0,0.0,8.0,6.0,3.0,0.0,8.0,0.0,3.0,9.0,0.0,7.0,9.0,2.0
            5.0,2.0,3.0,0.0,3.0,5.0,5.0,0.0,2.0,7.0,6.0,3.0,7.0,8.0,4.0,4.0
            7.0,8.0,3.0,3.0,0.0,5.0,2.0,0.0,8.0,7.0,7.0,1.0,6.0,5.0,8.0,3.0
            5.0,7.0,3.0,2.0,0.0,7.0,8.0,7.0,3.0,5.0,3.0,0.0,0.0,3.0,5.0,2.0
            1.0,6.0,0.0,3.0,5.0,6.0,9.0,7.0,7.0,8.0,2.0,1.0,7.0,0.0,6.0,1.0
            2.0,6.0,2.0,4.0,1.0,4.0,8.0,7.0,1.0,8.0,6.0,9.0,2.0,9.0,9.0,9.0
            8.0,7.0,9.0,0.0,7.0,0.0,9.0,9.0,3.0,6.0,8.0,8.0,8.0,5.0,4.0,0.0
            8.0,4.0,5.0,1.0,8.0,0.0,0.0,0.0,4.0,1.0,1.0,7.0,2.0,7.0,9.0,3.0
            5.0,8.0,7.0,2.0,2.0,6.0,8.0,8.0,0.0,1.0,1.0,2.0,6.0,8.0,6.0,7.0
            6.0,8.0,8.0,8.0,1.0,9.0,9.0,1.0,5.0,3.0,3.0,2.0,8.0,0.0,2.0,1.0
            6.0,2.0,6.0,9.0,2.0,0.0,4.0,7.0,1.0,2.0,5.0,7.0,7.0,9.0,8.0,2.0
            3.0,7.0,7.0,4.0,7.0,4.0,6.0,4.0,4.0,2.0,9.0,1.0,6.0,1.0,1.0,6.0
            8.0,8.0,7.0,1.0,5.0,6.0,2.0,3.0,8.0,1.0,2.0,3.0,1.0,9.0,9.0,9.0
            6.0,2.0,7.0,8.0,4.0,0.0,9.0,3.0,6.0,7.0,0.0,6.0,1.0,8.0,2.0,2.0
            1.0,5.0,4.0,6.0,2.0,4.0,6.0,8.0,2.0,3.0,7.0,3.0,3.0,7.0,6.0,8.0

    ];

    float input_B = [
            0.0,5.0,7.0,6.0,9.0,8.0,3.0,3.0,8.0,9.0,3.0,9.0,6.0,8.0,1.0,4.0
            7.0,8.0,3.0,5.0,2.0,6.0,5.0,9.0,4.0,1.0,4.0,8.0,1.0,8.0,5.0,9.0
            9.0,5.0,7.0,9.0,5.0,1.0,0.0,1.0,5.0,0.0,3.0,2.0,2.0,3.0,5.0,8.0
            9.0,3.0,6.0,1.0,9.0,9.0,1.0,6.0,3.0,5.0,9.0,2.0,8.0,5.0,6.0,4.0
            7.0,6.0,7.0,2.0,9.0,8.0,4.0,6.0,2.0,3.0,1.0,3.0,9.0,7.0,5.0,9.0
            8.0,6.0,1.0,4.0,5.0,2.0,8.0,0.0,8.0,8.0,6.0,8.0,9.0,0.0,3.0,6.0
            9.0,6.0,3.0,9.0,7.0,3.0,9.0,5.0,0.0,5.0,2.0,2.0,9.0,3.0,1.0,3.0
            6.0,9.0,8.0,6.0,2.0,7.0,9.0,0.0,8.0,0.0,5.0,2.0,5.0,1.0,6.0,3.0
            6.0,0.0,3.0,4.0,0.0,5.0,1.0,3.0,1.0,2.0,1.0,0.0,2.0,0.0,3.0,2.0
            9.0,7.0,3.0,1.0,7.0,1.0,7.0,0.0,5.0,0.0,7.0,9.0,3.0,7.0,3.0,7.0
            6.0,5.0,0.0,4.0,7.0,8.0,9.0,1.0,0.0,9.0,9.0,2.0,5.0,6.0,5.0,9.0
            0.0,9.0,7.0,7.0,3.0,5.0,5.0,8.0,2.0,2.0,6.0,3.0,6.0,4.0,8.0,3.0
            4.0,5.0,4.0,3.0,0.0,3.0,4.0,0.0,7.0,5.0,7.0,3.0,9.0,8.0,0.0,9.0
            0.0,9.0,9.0,2.0,2.0,9.0,9.0,6.0,6.0,0.0,9.0,4.0,2.0,3.0,9.0,5.0
            3.0,5.0,8.0,9.0,0.0,5.0,4.0,0.0,7.0,6.0,7.0,8.0,3.0,0.0,9.0,7.0
            5.0,5.0,4.0,2.0,6.0,9.0,1.0,2.0,3.0,2.0,8.0,5.0,2.0,1.0,7.0,8.0
    ];

    float *output_C = (float *) malloc(matSize);
    for (int i = 0; i < N2; i++)
        output_C[i] = 0;

    // malloc for cuda
    float *cuda_A = nullptr;
    hipMalloc(&cuda_A, matSize);

    float *cuda_B = nullptr;
    hipMalloc(&cuda_B, matSize);

    half *A = nullptr;
    hipMalloc(&A, matSize/2);

    half *B = nullptr;
    hipMalloc(&B, matSize/2);

    float *cuda_C = nullptr
    hipMalloc(&cuda_C, matSize);

    // copy to cuda
    hipMemcpy(cuda_A, input_A, matSize, hipMemcpyHostToDevice);
    hipMemcpy(cuda_B, input_B, matSize, hipMemcpyHostToDevice);

    float2half_mat<<<1, 16*16>>>(cuda_A, cuda_B, N2, A, B);

    warpGemm<<<1, 32>>>(A, B, cuda_C);

    hipMemcpy(output_C, cuda_C, matSize, hipMemcpyDeviceToHost);

    for(int i = 0; i < 16; i++)
        for (int j =0; j < 16; j++)
            printf("%f", output_C[i*16+j]);

    return 0;
}
